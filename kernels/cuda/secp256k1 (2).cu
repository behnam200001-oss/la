
#include <hip/hip_runtime.h>
#include <cstdint>

__device__ uint32_t rotr32(uint32_t x, int n) {
    return (x >> n) | (x << (32 - n));
}

__device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (~x & z);
}

__device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) ^ (x & z) ^ (y & z);
}

__device__ uint32_t sigma0(uint32_t x) {
    return rotr32(x, 2) ^ rotr32(x, 13) ^ rotr32(x, 22);
}

__device__ uint32_t sigma1(uint32_t x) {
    return rotr32(x, 6) ^ rotr32(x, 11) ^ rotr32(x, 25);
}

__device__ uint32_t gamma0(uint32_t x) {
    return rotr32(x, 7) ^ rotr32(x, 18) ^ (x >> 3);
}

__device__ uint32_t gamma1(uint32_t x) {
    return rotr32(x, 17) ^ rotr32(x, 19) ^ (x >> 10);
}

__constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

extern "C" __global__ void sha256_hash(
    const uint8_t* input,
    uint32_t input_len,
    uint8_t* output,
    uint32_t batch_size
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= batch_size) {
        return;
    }
    
    const uint8_t* data = input + idx * input_len;
    uint8_t* out = output + idx * 32;
    
    uint32_t h[8] = {
        0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
        0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
    };
    
    uint32_t total_bits = input_len * 8;
    uint32_t padding_len = (64 - ((input_len + 9) % 64)) % 64;
    uint32_t total_len = input_len + padding_len + 9;
    uint32_t num_blocks = total_len / 64;
    
    for (uint32_t block = 0; block < num_blocks; block++) {
        uint32_t w[64];
        
        for (int i = 0; i < 16; i++) {
            w[i] = 0;
            for (int j = 0; j < 4; j++) {
                uint32_t pos = block * 64 + i * 4 + j;
                if (pos < input_len) {
                    w[i] |= (uint32_t)data[pos] << (24 - j * 8);
                } else if (pos == input_len) {
                    w[i] |= 0x80 << (24 - j * 8);
                } else if (pos >= total_len - 8) {
                    w[i] |= (total_bits >> (56 - j * 8)) & 0xFF;
                }
            }
        }
        
        for (int i = 16; i < 64; i++) {
            w[i] = gamma1(w[i-2]) + w[i-7] + gamma0(w[i-15]) + w[i-16];
        }
        
        uint32_t a = h[0];
        uint32_t b = h[1];
        uint32_t c = h[2];
        uint32_t d = h[3];
        uint32_t e = h[4];
        uint32_t f = h[5];
        uint32_t g = h[6];
        uint32_t h_val = h[7];
        
        for (int i = 0; i < 64; i++) {
            uint32_t t1 = h_val + sigma1(e) + ch(e, f, g) + k[i] + w[i];
            uint32_t t2 = sigma0(a) + maj(a, b, c);
            h_val = g;
            g = f;
            f = e;
            e = d + t1;
            d = c;
            c = b;
            b = a;
            a = t1 + t2;
        }
        
        h[0] += a;
        h[1] += b;
        h[2] += c;
        h[3] += d;
        h[4] += e;
        h[5] += f;
        h[6] += g;
        h[7] += h_val;
    }
    
    for (int i = 0; i < 8; i++) {
        out[i*4] = (h[i] >> 24) & 0xFF;
        out[i*4+1] = (h[i] >> 16) & 0xFF;
        out[i*4+2] = (h[i] >> 8) & 0xFF;
        out[i*4+3] = h[i] & 0xFF;
    }
}